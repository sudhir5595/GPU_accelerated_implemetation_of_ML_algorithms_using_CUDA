#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>


// Kernel for GPU cuda 
__global__ void rand_dev(float* x, float* y, float* w_all, int k_value, int no_of_parameters, int n){
	int myid = threadIdx.x; 
	int block_id = blockIdx.x;
	float gradient[2]= {0};		
	double alpha[2];	//learning rate
	float fx, error =0.0;

	w_all[myid*no_of_parameters + 0] = 1;
	w_all[myid*no_of_parameters + 1] =-1; 

	alpha[0] = 0.035+0.01*(float)block_id;
	alpha[1] = 0.035+0.01*(float)block_id;

	//Train Loop
	for(int i=0; i<7000; i++){
	 	for(int k=0; k<n; k++){
			if (k< myid*(int)(n/k_value) || k >=((myid+1)*(int)(n/k_value))){
			fx=w_all[myid*no_of_parameters + 0]*x[k] + w_all[myid*no_of_parameters + 1];
			fx= 1/(1+exp(-fx));
			gradient[0]+= (y[k]-fx)*fx*(1-fx)*x[k];
			gradient[1]+= (y[k]-fx)*fx*(1-fx);
			}

		}
	w_all[myid*no_of_parameters + 0] += alpha[0]*(gradient[0]/900);
	w_all[myid*no_of_parameters + 1] += alpha[1]*(gradient[1]/900);
	gradient[0] = gradient[1] = 0.0;
	}

	//Testing loop
	for(int k=myid*(int)(n/k_value); k<(myid+1)*(int)(n/k_value); k++){
		fx = w_all[myid*no_of_parameters + 0]*x[k] + w_all[myid*no_of_parameters + 1];
		fx= 1/(1+exp(-fx));			
		error += (fx-y[k]);
	}
	printf("Learning rates: %.3f and %.3f---Weights of batch %d are: %.3f & %.3f---error is: %.3f\n", alpha[0], alpha[1], myid+1, w_all[myid*no_of_parameters + 0], w_all[myid*no_of_parameters + 1] , error);
}


int main(int argc, char *argv[]){
	int n=1000;// n is number of sample points
	int k_value =10, no_of_parameters=2;  
	float host_x[n], host_y[n], host_w_all[k_value][no_of_parameters]={0}, *x, *y, *w_all;
	float size = n * sizeof(float);	
	float size1 = k_value*no_of_parameters*sizeof(float);
	float a=0;

	//Data Generation with noise
	for(int i=0; i<n; i++){
		host_x[i] = a;
		a = a+0.05;
		float h = 0.18*host_x[i] -5.5; // m = 0.18 and c = -5.5
		host_y[i]=round(1/(1+exp(-h))); 
	}

	//Initializing the weights
	for(int i=0; i<k_value; i++){
		host_w_all[i][0]=1;
		host_w_all[i][1]=-1;
	}

	// allocating memory on cuda device
	hipMalloc(&x, size);
	hipMemcpy(x, host_x, size, hipMemcpyHostToDevice);
	hipMalloc(&y, size);
	hipMemcpy(y, host_y, size, hipMemcpyHostToDevice);
	hipMalloc(&w_all, size1);
	hipMemcpy(w_all,host_w_all, size1, hipMemcpyHostToDevice);

	dim3   DimGrid(5,1);     
	dim3   DimBlock(k_value,1);   
	rand_dev<<< DimGrid,DimBlock >>>(x, y, w_all, k_value, no_of_parameters, n);

	hipMemcpy(host_w_all, w_all, k_value, hipMemcpyDeviceToHost);
	hipFree(x);
	hipFree(y);
	hipFree(w_all);
}