#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>


// Kernel for GPU cuda 
__global__ void rand_dev(float* x, float* y, float* w_all, int k_value, int no_of_parameters, int n){
	int myid = threadIdx.x; 
	int block_id = blockIdx.x;
	float gradient[2]= {0};		
	double alpha[2] = {0.0000001,0.01};	//learning rate
	float fx, error =0.0;


	alpha[0] = 0.0000001*(block_id+1);
	alpha[1] = 0.01*(block_id+1);
	
	//Training Loop
	for(int epoch=0; epoch<5000; epoch++){
		for(int k=0; k<n; k++){
			if (k< myid*(int)(n/k_value) || k >=((myid+1)*(int)(n/k_value))){
				fx=w_all[myid*no_of_parameters + 0]*x[k] + w_all[myid*no_of_parameters + 1];
				gradient[0]+= (fx-y[k])*x[k];
				gradient[1]+= (fx-y[k]);
			}
		}
		w_all[myid*no_of_parameters + 0] -= alpha[0]*(gradient[0]/900);
		w_all[myid*no_of_parameters + 1] -= alpha[1]*(gradient[1]/900);
		gradient[0] = gradient[1] = 0.0;
	}

	//Testing Loop
	for(int k=myid*(int)(n/k_value); k<(myid+1)*(int)(n/k_value); k++){
		fx = w_all[myid*no_of_parameters + 0]*x[k] + w_all[myid*no_of_parameters + 1];
		error += (fx-y[k]);
	}
	printf("Learning rates: %.7f and %.3f---Weights of batch %d are: %.3f & %.3f---error is: %.3f\n", alpha[0], alpha[1], myid+1, w_all[myid*no_of_parameters + 0], w_all[myid*no_of_parameters + 1] , error);
}


int main(int argc, char *argv[]){

	clock_t t1, t2;
	int n=5000;// n is number of sample points
	int k_value =10, no_of_parameters=2;  
	float host_x[n], host_y[n], host_w_all[k_value][no_of_parameters]={0}, *x, *y, *w_all;
	float size = n * sizeof(float);	
	float size1 = k_value*no_of_parameters*sizeof(float);

	//Data Generation with noise
	for(int i=0; i<n; i++){
		host_x[i] = i;
		host_y[i] = host_x[i]*2 + 10; // m=2 and c=10
		host_y[i] += (float)rand()/RAND_MAX;
	}

	//Initializing the weights
	for(int i=0; i<k_value; i++)
		host_w_all[i][0]=1.5;

	// allocating memory on cuda device
	hipMalloc(&x, size);
	hipMemcpy(x, host_x, size, hipMemcpyHostToDevice);
	hipMalloc(&y, size);
	hipMemcpy(y, host_y, size, hipMemcpyHostToDevice);
	hipMalloc(&w_all, size1);
	hipMemcpy(w_all,host_w_all, size1, hipMemcpyHostToDevice);		


	dim3   DimGrid(5,1);     
	dim3   DimBlock(k_value,1);   

	t1=clock();

	rand_dev<<< DimGrid,DimBlock >>>(x, y, w_all, k_value, no_of_parameters, n);

	t2=clock();

	printf("#................................................................#");
	printf("\n Time taken for multiplication is %1fsec\n",(t2-t1)/(double) CLOCKS_PER_SEC);
	printf("#................................................................#");


	hipMemcpy(host_w_all, w_all, k_value, hipMemcpyDeviceToHost);
	hipFree(x);
	hipFree(y);
	hipFree(w_all);
}
